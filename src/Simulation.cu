#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>   // CUDA runtime API
#include <> // Optional: threadIdx, blockIdx, etc.
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include "Planet.h"
#include "Simulation.cuh"
#include <cuda/barrier>
#include "vectorOps.cuh"
#include "CudaHelpers.h"

namespace cg = cooperative_groups;

__device__ int frameCounter = 0;

__global__ void simulationKernel(Planet* dPlanets, Planet* dNextPlanets, int numPlanets) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    cg::grid_group grid = cg::this_grid();
    vec3 acc = {0, 0, 0};
    double G = 6.67430e-11; // N*m^2/kg^2
    float dt = 5;        // seconds

    // Return any threads that aren't being used
    if (idx >= numPlanets) {
        return;
    }

    while (true) {
        // Using Euler's method currently
        // Step 1: Calculate acceleration from all other planets' positions and masses
        acc = {0, 0, 0};    // meters/second^2
        for (int i = 0; i < numPlanets; i++) {
            if (i == idx)
                continue;
            vec3 diff = dPlanets[i].pos - dPlanets[idx].pos;
            float dist2 = dot(diff, diff);
            float inv_r = rsqrtf(dist2);
            acc += diff * (float)(G * dPlanets[i].mass) * inv_r * inv_r * inv_r;
        }
        dNextPlanets[idx].pos = dPlanets[idx].pos + dPlanets[idx].vel * dt;
        dNextPlanets[idx].vel = dPlanets[idx].vel + acc * dt;

        // Step 2: Synchronize all threads across the grid (each thread is 1 planet), and swap our two buffers 
        grid.sync();
        if (idx == 0)
            frameCounter += 1;

        cuda::std::swap(dPlanets, dNextPlanets);
    }
}



void Simulation::getPlanetsFromGPU()
{
    // Must launch it as an async request from a separate stream to not block and wait for the kernel to finish
    hipMemcpyAsync(m_hPlanets, m_dPlanets, m_numPlanets * sizeof(Planet), hipMemcpyDeviceToHost, m_dataStream);
    hipStreamSynchronize(m_dataStream);
    CudaHelpers::checkCudaErrors();
}

int Simulation::getFrameCountFromGPU()
{
    int hostValue;
    hipMemcpyFromSymbolAsync(&hostValue, HIP_SYMBOL(frameCounter), sizeof(int), 0, hipMemcpyDeviceToHost, m_dataStream);
    hipStreamSynchronize(m_dataStream);
    CudaHelpers::checkCudaErrors();
    return hostValue;
}

void Simulation::addPlanet(vec3 pos, vec3 vel, float mass)
{
    m_planets.push_back({pos, vel, mass});
}

Simulation::Simulation()
{
    // Initialize our arguments for the kernel
    m_args = (void **)malloc(sizeof(void*) * 3);
    m_args[0] = &m_hPlanets;
    m_args[1] = &m_dPlanets;
    m_args[2] = &m_numPlanets;
    
    hipStreamCreate(&m_kernelStream);
    hipStreamCreate(&m_dataStream);
    CudaHelpers::checkCudaErrors();
}

Simulation::~Simulation()
{
    free(m_args);
    hipStreamDestroy(m_kernelStream);
    hipStreamDestroy(m_dataStream);
    hipFree(m_dPlanets);
    hipFree(m_dNextPlanets);
    hipHostFree(m_hPlanets);
}

void Simulation::runSimulation()
{
    m_numPlanets = m_planets.size();

    const int threadsPerBlock = 128;
    const int blocks = m_numPlanets / (threadsPerBlock + 1) + 1;

    // Initialize variables on the host (CPU)
    hipHostAlloc((void **)(&m_hPlanets), m_numPlanets * sizeof(Planet), hipHostMallocDefault);
    CudaHelpers::checkCudaErrors();
    for (int i = 0; i < m_numPlanets; i++) {
        m_hPlanets[i] = m_planets[i];
    }
        
    // Initialize and allocate memory on the device (GPU)
    hipMalloc((void **)(&m_dPlanets), m_numPlanets * sizeof(Planet));
    hipMalloc((void **)(&m_dNextPlanets), m_numPlanets * sizeof(Planet));
    CudaHelpers::checkCudaErrors();

    // Copy values over to the device from the host
    hipMemcpy(m_dPlanets, m_hPlanets, m_numPlanets * sizeof(Planet), hipMemcpyHostToDevice);
    hipMemcpy(m_dNextPlanets, m_hPlanets, m_numPlanets * sizeof(Planet), hipMemcpyHostToDevice);
    CudaHelpers::checkCudaErrors();

    // Launch the kernel as cooperative so we can use grid.sync(), launch in a separate stream to be async
    std::cout << "Launching Kernel" << std::endl;
    hipLaunchCooperativeKernel((void*)simulationKernel, blocks, threadsPerBlock, m_args, 0, m_kernelStream);
    std::cout << "After launch" << std::endl;
    CudaHelpers::checkCudaErrors();
}
